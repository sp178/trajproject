#include "hip/hip_runtime.h"
#include"cudafunc.h"
#define check(real,want) {if(want!=real) return -100;}
#define Mass	 36000
#define Sref	 156.0771
#define Lref	 20.29
__device__ BLOCK<2>* cuCD = nullptr;
__device__ BLOCK<2>* cuCL = nullptr;
__device__ BLOCK<2>* cuCZ = nullptr;
__device__ BLOCK<1>* curho = nullptr;
__device__ BLOCK<1>* cuma = nullptr;

__device__ InterRecod<2>* cuCDrecord[BLOCK_NUM] ;
__device__ InterRecod<2>* cuCLrecord[BLOCK_NUM] ;
__device__ InterRecod<2>* cuCZrecord[BLOCK_NUM] ;
__device__ InterRecod<1>* curhorecord;
__device__ InterRecod<1>* cumarecord;


__device__ int TrajToryConst3D(spfloat _time, const spfloat* _x, spfloat* _f, spfloat* _param, uint32_t _index)
{
	interPlatheData	interdata;
	spfloat r = _x[0], miu = _x[1], lambda = _x[2], V = _x[3], gamma = _x[4], chi = _x[5];
	spfloat B, Re;
	spfloat sigma;
	spfloat graveV, graveT;
	spfloat P;
	spfloat R[3];
	spfloat angleofdirection;			//初始发射方位角和航迹偏角夹角
	spfloat SINMIU, COSMIU, SINGAMMA, COSGAMMA, SINCHI, COSCHI, SINSIGMA, COSSIGMA;
	//spfloat	COSGAMMALOCAL;
	spfloat POWRA_R;
	POWRA_R = pow(Constant_Earth_RA / r, 2);
	SINMIU = sin(miu);
	COSMIU = cos(miu);
	SINGAMMA = sin(gamma);
	COSGAMMA = cos(gamma);
	SINCHI = sin(chi);
	COSCHI = cos(chi);
	graveV = Constant_Earth_GM / (r * r) * (1 - 1.5*Constant_Earth_J2*POWRA_R*(5 * SINMIU*SINMIU - 1));
	graveT = 3 * Constant_Earth_J2*Constant_Earth_GM / (r*r)*POWRA_R;
	B = atan2(SINMIU / COSMIU, Constant_Earth_e2);
	Re = Constant_Earth_RA*Constant_Earth_e / sqrt((SINMIU*SINMIU + Constant_Earth_e2*COSMIU*COSMIU));
	interdata.h = sqrt(r*r - Re*Re*pow(sin(B - miu), 2)) - Re*cos(B - miu);
	interdata.alpha = getAlpha(V);
	//COSGAMMALOCAL = cos(gamma + length(_param[3], _param[4], lambda, miu));
	uint32_t block = _index / THREAD_NUM;
	uint32_t thread = _index % THREAD_NUM;
	P = 0.5*interplate(curho, &interdata.h, curhorecord + _index)*Sref*V*V;
	interdata.Ma = V / interplate(cuma, &interdata.h, cumarecord + _index);
	R[0] = -P*interplate(cuCD, &interdata.alpha, cuCDrecord[block] + thread);
	R[1] = P*interplate(cuCL, &interdata.alpha, cuCLrecord[block] + thread);
	R[2] = P*interplate(cuCZ, &interdata.alpha, cuCZrecord[block] + thread);


	//	sigmamin = (GM / (r*r) - V*V / r)*Mass / R[1];
	sigma = _param[0];
	SINSIGMA = sin(sigma);
	COSSIGMA = cos(sigma);
	angleofdirection = chi - _param[1];
	_f[0] = V*SINGAMMA;
	_f[1] = V*COSGAMMA*COSCHI / r;
	_f[2] = V*COSGAMMA*SINCHI / (r*COSMIU);
	_f[3] = -R[0] / Mass - graveV *SINGAMMA \
		- graveT* SINMIU*(COSCHI*COSGAMMA*COSMIU + SINGAMMA*SINMIU)\
		- Constant_Earth_We*Constant_Earth_We * r*COSMIU*(COSCHI*COSGAMMA*SINMIU - SINGAMMA*COSMIU);
	_f[4] = R[1] / (Mass*V)*COSSIGMA - R[2] / (Mass*V)*SINGAMMA\
		- graveV *COSGAMMA / V\
		+ graveT * SINMIU*(COSCHI*SINGAMMA*COSMIU - COSGAMMA*SINMIU) / V\
		+ V*COSGAMMA / r;
	_f[5] = (-R[2] * COSSIGMA - R[1] * SINSIGMA + graveT * Mass*SINMIU*COSMIU*SINCHI \
		+ Mass*V*V * SINMIU / COSMIU*COSGAMMA*COSGAMMA* SINCHI / r) / (Mass*V*COSGAMMA);
	_f[6] = -V*COSGAMMA*cos(angleofdirection) / r;
	if ((interdata.h*Constant_Earth_g0 + 0.5*V*V) > (HIGHTMIN*Constant_Earth_g0 + 0.5*SPEEDEND*SPEEDEND)) return 0;
	else return 1;
}




__device__ int TrajTory3D(spfloat _time, const spfloat* _x, spfloat* _f, spfloat* _param, uint32_t _index)
{
	interPlatheData	interdata;
	spfloat r = _x[0], miu = _x[1], lambda = _x[2], V = _x[3], gamma = _x[4], chi = _x[5];
	spfloat B, Re;
	spfloat sigma;
	spfloat graveV, graveT;
	spfloat P;
	spfloat R[3];
	spfloat angleofdirection;			//初始发射方位角和航迹偏角夹角
	spfloat SINMIU, COSMIU, SINGAMMA, COSGAMMA, SINCHI, COSCHI, SINSIGMA, COSSIGMA;
	//spfloat	COSGAMMALOCAL;
	spfloat POWRA_R;
	POWRA_R = pow(Constant_Earth_RA / r, 2);
	SINMIU = sin(miu);
	COSMIU = cos(miu);
	SINGAMMA = sin(gamma);
	COSGAMMA = cos(gamma);
	SINCHI = sin(chi);
	COSCHI = cos(chi);
	graveV = Constant_Earth_GM / (r * r) * (1 - 1.5*Constant_Earth_J2*POWRA_R*(5 * SINMIU*SINMIU - 1));
	graveT = 3 * Constant_Earth_J2*Constant_Earth_GM / (r*r)*POWRA_R;
	B = atan2(SINMIU / COSMIU, Constant_Earth_e2);
	Re = Constant_Earth_RA*Constant_Earth_e / sqrt((SINMIU*SINMIU + Constant_Earth_e2*COSMIU*COSMIU));
	interdata.h = sqrt(r*r - Re*Re*pow(sin(B - miu), 2)) - Re*cos(B - miu);
	interdata.alpha = getAlpha(V);
	//COSGAMMALOCAL = cos(gamma + length(_param[3], _param[4], lambda, miu));
	uint32_t block = _index / THREAD_NUM;
	uint32_t thread = _index % THREAD_NUM;
	P = 0.5*interplate(curho, &interdata.h, curhorecord + _index)*Sref*V*V;
	interdata.Ma = V / interplate(cuma, &interdata.h, cumarecord + _index);
	R[0] = -P*interplate(cuCD, &interdata.alpha, cuCDrecord[block] + thread);
	R[1] = P*interplate(cuCL, &interdata.alpha, cuCLrecord[block] + thread);
	R[2] = P*interplate(cuCZ, &interdata.alpha, cuCZrecord[block] + thread);


	//	sigmamin = (GM / (r*r) - V*V / r)*Mass / R[1];
	sigma = getSigma(V, _param[0], _param[2]);
	//sigma = _param[0];
	SINSIGMA = sin(sigma);
	COSSIGMA = cos(sigma);
	angleofdirection = chi - _param[1];
	_f[0] = V*SINGAMMA;
	_f[1] = V*COSGAMMA*COSCHI / r;
	_f[2] = V*COSGAMMA*SINCHI / (r*COSMIU);
	_f[3] = -R[0] / Mass - graveV *SINGAMMA \
		- graveT* SINMIU*(COSCHI*COSGAMMA*COSMIU + SINGAMMA*SINMIU)\
		- Constant_Earth_We*Constant_Earth_We * r*COSMIU*(COSCHI*COSGAMMA*SINMIU - SINGAMMA*COSMIU);
	_f[4] = R[1] / (Mass*V)*COSSIGMA - R[2] / (Mass*V)*SINGAMMA\
		- graveV *COSGAMMA / V\
		+ graveT * SINMIU*(COSCHI*SINGAMMA*COSMIU - COSGAMMA*SINMIU) / V\
		+ V*COSGAMMA / r;
	_f[5] = (-R[2] * COSSIGMA - R[1] * SINSIGMA + graveT * Mass*SINMIU*COSMIU*SINCHI \
		+ Mass*V*V * SINMIU / COSMIU*COSGAMMA*COSGAMMA* SINCHI / r) / (Mass*V*COSGAMMA);
	_f[6] = -V*COSGAMMA*cos(angleofdirection) / r;
	if ((interdata.h*Constant_Earth_g0 + 0.5*V*V) > (HIGHTMIN*Constant_Earth_g0 + 0.5*SPEEDEND*SPEEDEND)) return 0;
	else return 1;
}
__device__ int TrajOut(spfloat _time, const spfloat* _x, spfloat* _param, uint32_t _index)
{

	if (fabs(_time - RECORDTIME) < 0.01)
	{
		spfloat r = _x[0], miu = _x[1], V = _x[3];
		spfloat h;
		spfloat B, Re;
		spfloat SINMIU, COSMIU;
		SINMIU = sin(miu);
		COSMIU = cos(miu);
		B = atan2(SINMIU / COSMIU, Constant_Earth_e2);
		Re = Constant_Earth_RA*Constant_Earth_e / sqrt((SINMIU*SINMIU + Constant_Earth_e2*COSMIU*COSMIU));
		h = sqrt(r*r - Re*Re*pow(sin(B - miu), 2)) - Re*cos(B - miu);
		_param[5] = 0.5*V*V + h*Constant_Earth_g0;
		_param[6] = _x[5] - _param[1];
	}
	return 0;
}
__global__ void InitialData(BLOCK<2>* _cuCD1,
	BLOCK<2>* _cuCL1,
	BLOCK<2>* _cuCZ1,
	BLOCK<1>* _curho1,
	BLOCK<1>* _cuma1)
{
	size_t _index1 = threadIdx.x + blockIdx.x*blockDim.x;
	if (0 == _index1)
	{
		cuCD = _cuCD1;
		cuCL = _cuCL1;
		cuCZ = _cuCZ1;
		curho = _curho1;
		cuma = _cuma1;
//		cuCDrecord = (InterRecod<2>*) malloc(THREAD_NUM*BLOCK_NUM * sizeof(InterRecod<2>));
//		cuCLrecord = (InterRecod<2>*) malloc(THREAD_NUM*BLOCK_NUM * sizeof(InterRecod<2>));
//		cuCZrecord = (InterRecod<2>*) malloc(THREAD_NUM*BLOCK_NUM * sizeof(InterRecod<2>));
		curhorecord = (InterRecod<1>*) malloc(THREAD_NUM*BLOCK_NUM * sizeof(InterRecod<1>));
		cumarecord = (InterRecod<1>*) malloc(THREAD_NUM*BLOCK_NUM * sizeof(InterRecod<1>));
	}
	__syncthreads();
};
__global__ void TrajtoryGPU(rk4_state* m_rkdata,
							Traj* _begTraj, 
							spfloat* _tarjet,
							spfloat _step, 
							spfloat _sigma, 
							spfloat _searchangle)
{
	__shared__ double sharedata_[(CUSTATANUM * 4 + CUOUTNUM+3*3)* THREAD_NUM];
	size_t _index1 = threadIdx.x + blockIdx.x*blockDim.x;
	double *data = sharedata_ + threadIdx.x*(CUSTATANUM * 4 + CUOUTNUM);

	if (_index1 >= THREAD_NUM*BLOCK_NUM) return;
	if (1 == threadIdx.x)
	{
		cuCDrecord[blockIdx.x] = (InterRecod<2>*)(sharedata_ + (CUSTATANUM * 4 + CUOUTNUM)*THREAD_NUM);
		cuCLrecord[blockIdx.x] = (InterRecod<2>*)(sharedata_ + (CUSTATANUM * 4 + CUOUTNUM+3)* THREAD_NUM);
		cuCZrecord[blockIdx.x] = (InterRecod<2>*)(sharedata_ + (CUSTATANUM * 4 + CUOUTNUM+3*2) * THREAD_NUM );
		//curhorecord[blockIdx.x] = (InterRecod<1>*)(sharedata_ + THREAD_NUM * 3 * 3);
		//cumarecord[blockIdx.x] = (InterRecod<1>*)(sharedata_ + THREAD_NUM * 3 * 4);
	}
	m_rkdata[_index1].x		=  data ;
	m_rkdata[_index1].k		=  data +CUSTATANUM;
	m_rkdata[_index1].x0	=  data +2 * CUSTATANUM;
	m_rkdata[_index1].xtmp  =  data +3 * CUSTATANUM;
	m_rkdata[_index1].param =  data  +4 * CUSTATANUM;
	memcpy(m_rkdata[_index1].x0, _begTraj, sizeof(spfloat)*CUSTATANUM);
	//m_rkdata[_index1].param[0] = deg2rad(-85);
	//m_rkdata[_index1].param[2] = 3000;
	m_rkdata[_index1].param[0] = _sigma + spfloat(_index1)*_searchangle / gridDim.x / blockDim.x;	//起始角度
	m_rkdata[_index1].param[1] = _begTraj->lanuchangle;		//内部参数 方位角
	m_rkdata[_index1].param[2] = _begTraj->V;		//起始速度
	m_rkdata[_index1].param[3] = _begTraj->begLongt;		//起始经度
	m_rkdata[_index1].param[4] = _begTraj->begLat;		//起始纬度
	m_rkdata[_index1].isDown = CalCulate(m_rkdata[_index1], _step, 0, _index1);
	__syncthreads();
	//航程差
	_tarjet[CUOUTNUM * _index1] = m_rkdata[_index1].x0[6];
	//记录纬度1
	_tarjet[CUOUTNUM * _index1 + 1] = m_rkdata[_index1].x0[1];
	//记录经度2
	_tarjet[CUOUTNUM * _index1 + 2] = m_rkdata[_index1].x0[2];
	//记录预测点和目标点距离3
	_tarjet[CUOUTNUM * _index1 + 3] = length(m_rkdata[_index1].x0[2], m_rkdata[_index1].x0[1], _begTraj->targetlambda, _begTraj->targetmiu);
	//横向误差4	
	_tarjet[CUOUTNUM * _index1 + 4] = getCrose(_begTraj->targetlambda, _begTraj->targetmiu, _begTraj->lambda, _begTraj->miu, m_rkdata[_index1].x0[2], m_rkdata[_index1].x0[1]);
	//能量5
	_tarjet[CUOUTNUM * _index1 + 5] = m_rkdata[_index1].param[5];
	//十秒后的弹道偏角
	_tarjet[CUOUTNUM * _index1 + 6] = m_rkdata[_index1].param[6];
	__syncthreads();
}; 

__global__ void InitialStata(rk4_state* m_rkdata,spfloat* _paramdata) {
	size_t _index1 = threadIdx.x + blockIdx.x*blockDim.x;

	//if (0 == _index1) {
	//	m_rkdata = (rk4_state*)malloc(BLOCK_NUM*THREAD_NUM * sizeof(rk4_state));
	//}
	//__syncthreads();
//  m_rkdata[_index1].x		=(spfloat*)malloc(sizeof(spfloat)*CUSTATANUM);
//  m_rkdata[_index1].k		=(spfloat*)malloc(sizeof(spfloat)*CUSTATANUM);
//  m_rkdata[_index1].x0	=(spfloat*)malloc(sizeof(spfloat)*CUSTATANUM);			//状态初始值
//  m_rkdata[_index1].xtmp	=(spfloat*)malloc(sizeof(spfloat)*CUSTATANUM);			//状态过渡值
//  m_rkdata[_index1].param = (spfloat*)malloc(sizeof(spfloat)*CUOUTNUM);;		//外部参数
	m_rkdata[_index1].dim = CUSTATANUM;							//7维积分
	m_rkdata[_index1].paramdim = 2;						//目标点经纬度
	m_rkdata[_index1].func = TrajTory3D;
	m_rkdata[_index1].funcOut = TrajOut;

};
__global__ void InitialStataConst(rk4_state* m_rkdata, spfloat* _paramdata) {
	size_t _index1 = threadIdx.x + blockIdx.x*blockDim.x;

	//if (0 == _index1) {
	//	m_rkdata = (rk4_state*)malloc(BLOCK_NUM*THREAD_NUM * sizeof(rk4_state));
	//}
	//__syncthreads();
	//  m_rkdata[_index1].x		=(spfloat*)malloc(sizeof(spfloat)*CUSTATANUM);
	//  m_rkdata[_index1].k		=(spfloat*)malloc(sizeof(spfloat)*CUSTATANUM);
	//  m_rkdata[_index1].x0	=(spfloat*)malloc(sizeof(spfloat)*CUSTATANUM);			//状态初始值
	//  m_rkdata[_index1].xtmp	=(spfloat*)malloc(sizeof(spfloat)*CUSTATANUM);			//状态过渡值
	//  m_rkdata[_index1].param = (spfloat*)malloc(sizeof(spfloat)*CUOUTNUM);;		//外部参数
	m_rkdata[_index1].dim = CUSTATANUM;							//7维积分
	m_rkdata[_index1].paramdim = 2;						//目标点经纬度
	m_rkdata[_index1].func = TrajToryConst3D;
	m_rkdata[_index1].funcOut = TrajOut;

};

int cudafunc::initialCudaData(BLOCK<2>* _CD, BLOCK<2>* _CL, BLOCK<2>* _CZ, BLOCK<1>* _rho, BLOCK<1>* _ma)
{
	if (isinitial) return 0;
	isinitial = true;
	hipError_t cuerror;
	double *decordials_, *debeg_;
	check(hipMalloc(&cuCD,sizeof(BLOCK<2>)), ::hipSuccess);
	check(hipMalloc(&cuCL, sizeof(BLOCK<2>)), ::hipSuccess);
	check(hipMalloc(&cuCZ, sizeof(BLOCK<2>)), ::hipSuccess);
	check(hipMalloc(&curho, sizeof(BLOCK<2>)), ::hipSuccess);
	check(hipMalloc(&cuma, sizeof(BLOCK<2>)), ::hipSuccess);
	check(hipMalloc(&cuTraj, sizeof(Traj)), ::hipSuccess);
	check(hipMalloc(&cuoutdata, sizeof(spfloat)*CUOUTNUM*BLOCK_NUM*THREAD_NUM), ::hipSuccess);
	size_t _dementionlength = 0;	//坐标维度
	size_t _datalength = 1;		//数据长度
	for (size_t index = 0; index<_CD->DIM; ++index)
	{
		_dementionlength += _CD->m_cordial_demtion[index];
		_datalength *= _CD->m_cordial_demtion[index];
	}

	if (nullptr != cuCD) {
		check(hipMalloc((void**)& decordials_, sizeof(spfloat)*_dementionlength), ::hipSuccess);
		check(hipMalloc((void**)& debeg_, sizeof(spfloat)*_datalength), ::hipSuccess);
		check(hipMemcpy(decordials_, _CD->m_cordials, sizeof(spfloat)*_dementionlength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(debeg_, _CD->m_beg, sizeof(spfloat)*_datalength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(cuCD->m_cordial_demtion, _CD->m_cordial_demtion, sizeof(uint32_t)*BLOCK<2>::DIM, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCD->m_beg), &debeg_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCD->m_cordials), &decordials_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		debeg_ = nullptr;
		decordials_=nullptr;
	}

	_dementionlength = 0;	//坐标维度
	_datalength = 1;		//数据长度
	for (size_t index = 0; index<_CL->DIM; ++index)
	{
		_dementionlength += _CL->m_cordial_demtion[index];
		_datalength *= _CL->m_cordial_demtion[index];
	}

	if (nullptr != cuCL) {
		check(hipMalloc((void**)& decordials_, sizeof(spfloat)*_dementionlength), ::hipSuccess);
		check(hipMalloc((void**)& debeg_, sizeof(spfloat)*_datalength), ::hipSuccess);
		check(hipMemcpy(decordials_, _CL->m_cordials, sizeof(spfloat)*_dementionlength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(debeg_, _CL->m_beg, sizeof(spfloat)*_datalength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(cuCL->m_cordial_demtion, _CL->m_cordial_demtion, sizeof(uint32_t)*BLOCK<2>::DIM, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCL->m_beg), &debeg_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCL->m_cordials), &decordials_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		debeg_ = nullptr;
		decordials_ = nullptr;
	}

	_dementionlength = 0;	//坐标维度
	_datalength = 1;		//数据长度
	for (size_t index = 0; index<_CZ->DIM; ++index)
	{
		_dementionlength += _CZ->m_cordial_demtion[index];
		_datalength *= _CZ->m_cordial_demtion[index];
	}

	if (nullptr != cuCZ) {
		check(hipMalloc((void**)& decordials_, sizeof(spfloat)*_dementionlength), ::hipSuccess);
		check(hipMalloc((void**)& debeg_, sizeof(spfloat)*_datalength), ::hipSuccess);
		check(hipMemcpy(decordials_, _CZ->m_cordials, sizeof(spfloat)*_dementionlength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(debeg_, _CZ->m_beg, sizeof(spfloat)*_datalength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(cuCZ->m_cordial_demtion, _CZ->m_cordial_demtion, sizeof(uint32_t)*BLOCK<2>::DIM, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCZ->m_beg), &debeg_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCZ->m_cordials), &decordials_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		debeg_ = nullptr;
		decordials_ = nullptr;
	}

	_dementionlength = 0;	//坐标维度
	_datalength = 1;		//数据长度
	for (size_t index = 0; index<_rho->DIM; ++index)
	{
		_dementionlength += _rho->m_cordial_demtion[index];
		_datalength *= _rho->m_cordial_demtion[index];
	}

	if (nullptr != curho) {
		check(hipMalloc((void**)& decordials_, sizeof(spfloat)*_dementionlength), ::hipSuccess);
		check(hipMalloc((void**)& debeg_, sizeof(spfloat)*_datalength), ::hipSuccess);
		check(hipMemcpy(decordials_, _rho->m_cordials, sizeof(spfloat)*_dementionlength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(debeg_, _rho->m_beg, sizeof(spfloat)*_datalength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(curho->m_cordial_demtion, _rho->m_cordial_demtion, sizeof(uint32_t)*BLOCK<2>::DIM, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(curho->m_beg), &debeg_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(curho->m_cordials), &decordials_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		debeg_ = nullptr;
		decordials_ = nullptr;
	}

	_dementionlength = 0;	//坐标维度
	_datalength = 1;		//数据长度
	for (size_t index = 0; index<_ma->DIM; ++index)
	{
		_dementionlength += _ma->m_cordial_demtion[index];
		_datalength *= _ma->m_cordial_demtion[index];
	}

	if (nullptr != cuma) {
		check(hipMalloc((void**)& decordials_, sizeof(spfloat)*_dementionlength), ::hipSuccess);
		check(hipMalloc((void**)& debeg_, sizeof(spfloat)*_datalength), ::hipSuccess);
		check(hipMemcpy(decordials_, _ma->m_cordials, sizeof(spfloat)*_dementionlength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(debeg_, _ma->m_beg, sizeof(spfloat)*_datalength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(cuma->m_cordial_demtion, _ma->m_cordial_demtion, sizeof(uint32_t)*BLOCK<2>::DIM, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuma->m_beg), &debeg_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuma->m_cordials), &decordials_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		debeg_ = nullptr;
		decordials_ = nullptr;
	}

	check(hipMalloc(&cuTraj, sizeof(Traj)),::hipSuccess);
	check(hipMalloc(&cuoutdata, sizeof(spfloat)*CUOUTNUM*BLOCK_NUM*THREAD_NUM),::hipSuccess);

	InitialData <<<1, 1 >>>(cuCD, cuCL, cuCZ, curho, cuma);

	check(hipMalloc(&m_rkdata,BLOCK_NUM*THREAD_NUM * sizeof(rk4_state)), ::hipSuccess);
	check(hipMalloc(&m_paramdata, BLOCK_NUM*THREAD_NUM *(CUOUTNUM+CUSTATANUM*4)*sizeof(spfloat)), ::hipSuccess);
	InitialStata <<<BLOCK_NUM, THREAD_NUM>>> (m_rkdata, m_paramdata);
	check(hipDeviceSynchronize(), ::hipSuccess);
	return 0;
}

int cudafunc::initialCudaDataConst(BLOCK<2>* _CD, BLOCK<2>* _CL, BLOCK<2>* _CZ, BLOCK<1>* _rho, BLOCK<1>* _ma)
{
	if (isinitial) return 0;
	isinitial = true;
	hipError_t cuerror;
	double *decordials_, *debeg_;
	check(hipMalloc(&cuCD, sizeof(BLOCK<2>)), ::hipSuccess);
	check(hipMalloc(&cuCL, sizeof(BLOCK<2>)), ::hipSuccess);
	check(hipMalloc(&cuCZ, sizeof(BLOCK<2>)), ::hipSuccess);
	check(hipMalloc(&curho, sizeof(BLOCK<2>)), ::hipSuccess);
	check(hipMalloc(&cuma, sizeof(BLOCK<2>)), ::hipSuccess);
	check(hipMalloc(&cuTraj, sizeof(Traj)), ::hipSuccess);
	check(hipMalloc(&cuoutdata, sizeof(spfloat)*CUOUTNUM*BLOCK_NUM*THREAD_NUM), ::hipSuccess);
	size_t _dementionlength = 0;	//坐标维度
	size_t _datalength = 1;		//数据长度
	for (size_t index = 0; index<_CD->DIM; ++index)
	{
		_dementionlength += _CD->m_cordial_demtion[index];
		_datalength *= _CD->m_cordial_demtion[index];
	}

	if (nullptr != cuCD) {
		check(hipMalloc((void**)& decordials_, sizeof(spfloat)*_dementionlength), ::hipSuccess);
		check(hipMalloc((void**)& debeg_, sizeof(spfloat)*_datalength), ::hipSuccess);
		check(hipMemcpy(decordials_, _CD->m_cordials, sizeof(spfloat)*_dementionlength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(debeg_, _CD->m_beg, sizeof(spfloat)*_datalength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(cuCD->m_cordial_demtion, _CD->m_cordial_demtion, sizeof(uint32_t)*BLOCK<2>::DIM, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCD->m_beg), &debeg_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCD->m_cordials), &decordials_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		debeg_ = nullptr;
		decordials_ = nullptr;
	}

	_dementionlength = 0;	//坐标维度
	_datalength = 1;		//数据长度
	for (size_t index = 0; index<_CL->DIM; ++index)
	{
		_dementionlength += _CL->m_cordial_demtion[index];
		_datalength *= _CL->m_cordial_demtion[index];
	}

	if (nullptr != cuCL) {
		check(hipMalloc((void**)& decordials_, sizeof(spfloat)*_dementionlength), ::hipSuccess);
		check(hipMalloc((void**)& debeg_, sizeof(spfloat)*_datalength), ::hipSuccess);
		check(hipMemcpy(decordials_, _CL->m_cordials, sizeof(spfloat)*_dementionlength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(debeg_, _CL->m_beg, sizeof(spfloat)*_datalength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(cuCL->m_cordial_demtion, _CL->m_cordial_demtion, sizeof(uint32_t)*BLOCK<2>::DIM, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCL->m_beg), &debeg_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCL->m_cordials), &decordials_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		debeg_ = nullptr;
		decordials_ = nullptr;
	}

	_dementionlength = 0;	//坐标维度
	_datalength = 1;		//数据长度
	for (size_t index = 0; index<_CZ->DIM; ++index)
	{
		_dementionlength += _CZ->m_cordial_demtion[index];
		_datalength *= _CZ->m_cordial_demtion[index];
	}

	if (nullptr != cuCZ) {
		check(hipMalloc((void**)& decordials_, sizeof(spfloat)*_dementionlength), ::hipSuccess);
		check(hipMalloc((void**)& debeg_, sizeof(spfloat)*_datalength), ::hipSuccess);
		check(hipMemcpy(decordials_, _CZ->m_cordials, sizeof(spfloat)*_dementionlength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(debeg_, _CZ->m_beg, sizeof(spfloat)*_datalength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(cuCZ->m_cordial_demtion, _CZ->m_cordial_demtion, sizeof(uint32_t)*BLOCK<2>::DIM, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCZ->m_beg), &debeg_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuCZ->m_cordials), &decordials_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		debeg_ = nullptr;
		decordials_ = nullptr;
	}

	_dementionlength = 0;	//坐标维度
	_datalength = 1;		//数据长度
	for (size_t index = 0; index<_rho->DIM; ++index)
	{
		_dementionlength += _rho->m_cordial_demtion[index];
		_datalength *= _rho->m_cordial_demtion[index];
	}

	if (nullptr != curho) {
		check(hipMalloc((void**)& decordials_, sizeof(spfloat)*_dementionlength), ::hipSuccess);
		check(hipMalloc((void**)& debeg_, sizeof(spfloat)*_datalength), ::hipSuccess);
		check(hipMemcpy(decordials_, _rho->m_cordials, sizeof(spfloat)*_dementionlength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(debeg_, _rho->m_beg, sizeof(spfloat)*_datalength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(curho->m_cordial_demtion, _rho->m_cordial_demtion, sizeof(uint32_t)*BLOCK<2>::DIM, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(curho->m_beg), &debeg_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(curho->m_cordials), &decordials_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		debeg_ = nullptr;
		decordials_ = nullptr;
	}

	_dementionlength = 0;	//坐标维度
	_datalength = 1;		//数据长度
	for (size_t index = 0; index<_ma->DIM; ++index)
	{
		_dementionlength += _ma->m_cordial_demtion[index];
		_datalength *= _ma->m_cordial_demtion[index];
	}

	if (nullptr != cuma) {
		check(hipMalloc((void**)& decordials_, sizeof(spfloat)*_dementionlength), ::hipSuccess);
		check(hipMalloc((void**)& debeg_, sizeof(spfloat)*_datalength), ::hipSuccess);
		check(hipMemcpy(decordials_, _ma->m_cordials, sizeof(spfloat)*_dementionlength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(debeg_, _ma->m_beg, sizeof(spfloat)*_datalength, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(cuma->m_cordial_demtion, _ma->m_cordial_demtion, sizeof(uint32_t)*BLOCK<2>::DIM, ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuma->m_beg), &debeg_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		check(hipMemcpy(&(cuma->m_cordials), &decordials_, sizeof(double*), ::hipMemcpyHostToDevice), ::hipSuccess);
		debeg_ = nullptr;
		decordials_ = nullptr;
	}

	check(hipMalloc(&cuTraj, sizeof(Traj)), ::hipSuccess);
	check(hipMalloc(&cuoutdata, sizeof(spfloat)*CUOUTNUM*BLOCK_NUM*THREAD_NUM), ::hipSuccess);

	InitialData << <1, 1 >> >(cuCD, cuCL, cuCZ, curho, cuma);

	check(hipMalloc(&m_rkdata, BLOCK_NUM*THREAD_NUM * sizeof(rk4_state)), ::hipSuccess);
	check(hipMalloc(&m_paramdata, BLOCK_NUM*THREAD_NUM *(CUOUTNUM + CUSTATANUM * 4) * sizeof(spfloat)), ::hipSuccess);
	InitialStataConst << <BLOCK_NUM, THREAD_NUM >> > (m_rkdata, m_paramdata);
	check(hipDeviceSynchronize(), ::hipSuccess);
	return 0;
}




int cudafunc::TrajCaclCUDA(Traj * _beg, spfloat * _targrt, spfloat _step, spfloat _sigma, spfloat _searchangle)
{
	hipError_t cudaStatus;
	check(hipMemcpy(cuTraj, _beg, sizeof(Traj), ::hipMemcpyHostToDevice), ::hipSuccess);
	TrajtoryGPU << <BLOCK_NUM, THREAD_NUM >> > (m_rkdata,cuTraj, cuoutdata, _step, _sigma, _searchangle);
	check(hipDeviceSynchronize(), ::hipSuccess);
	check(hipMemcpy(_targrt, cuoutdata, sizeof(spfloat)*CUOUTNUM*BLOCK_NUM*THREAD_NUM, ::hipMemcpyDeviceToHost), ::hipSuccess);
	return 0;
}