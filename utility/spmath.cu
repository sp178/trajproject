#include "hip/hip_runtime.h"
#include "spmath.h"

__device__ __host__ double sgn(double _data)
{
	return _data < 0 ? -1 : 1;
}
__device__ __host__ double length(double _longdest, double _ladest, double _longtsrc, double _latsrc)
{
	return acos(cos(_ladest) * cos(_latsrc) * cos(_longdest - _longtsrc) + sin(_ladest) * sin(_latsrc));
}
__device__ __host__ double lanuchAngle(double _longdest, double _ladest, double _longtsrc, double _latsrc)
{
	double a;
	double c = length(_longdest, _ladest, _longtsrc, _latsrc);
	//�ж��Ƿ����Ұ�ƽ��
	if (_longdest > _longtsrc)
	{
		if (_ladest > _latsrc)
		{ //��
			a = length(_longdest, _ladest, _longtsrc, _ladest);
			if (c < 1E-13)
				return 0;
			return asin(sin(a) / sin(c));
		}
		else
		{
			a = length(_longdest, _ladest, _longdest, _latsrc);
			if (c < 1E-13)
				return Constant_PI * 0.5;
			return Constant_PI * 0.5 + asin(sin(a) / sin(c));
		}
	}
	else
	{ //���ƽ��
		if (_ladest > _latsrc)
		{ //��
			a = length(_longdest, _ladest, _longtsrc, _ladest);
			if (c < 1E-13)
				return 0;
			return -asin(sin(a) / sin(c));
		}
		else
		{
			a = length(_longdest, _ladest, _longdest, _latsrc);
			if (c < 1E-13)
				return -Constant_PI;
			return -(Constant_PI + asin(sin(a) / sin(c)));
		}
	}
}

__device__ __host__ int rt4_step(
	const rk4_state &state,
	const spfloat h, const spfloat t, uint32_t _index);
/*
�ⲿ������ýӿ�
*/

/*����ÿһ�����*/
__device__ __host__ int CalOut(spfloat _time, const rk4_state &state, uint32_t _index)
{
	if (state.funcOut)
		return state.funcOut(_time, state.x0, state.param, _index);
	return 0;
}
__device__ __host__ int CalCulate(const rk4_state &state,
	const spfloat h, const spfloat t, uint32_t _index)
{
	spfloat time = 0, step = h;
	short _bastate;
	for (;;)
	{
		_bastate = CalOut(time, state, _index);
		if (time > 1000)
			_bastate = 1;
		CHACK_RETURN0(_bastate);
		_bastate = rt4_step(state, step, time, _index);
		time += step;
		CHACK_RETURN0(_bastate);
	}
}

/*
���º���ȫ������cudaʹ�� ��ͨ������ý�ֹ
*/

__device__ __host__ double altitude(double _lat, double _r)
{
	double SINMIU, COSMIU, B, Re, h;
	SINMIU = sin(_lat);
	COSMIU = cos(_lat);
	B = atan2(SINMIU / COSMIU, e2);
	Re = R_a * e / sqrt((SINMIU * SINMIU + e2 * COSMIU * COSMIU));
	h = sqrt(_r * _r - Re * Re * pow(sin(B - _lat), 2)) - Re * cos(B - _lat);
	return h;
}

__device__ __host__ int rt4_step(
	const rk4_state &state,
	const spfloat h, const spfloat t, uint32_t _index)
{
	spfloat *x = state.x; //״̬����
	spfloat *k = state.k;
	spfloat *x0 = state.x0;         //״̬��ʼֵ
	spfloat *xtmp = state.xtmp;     //״̬����ֵ
	spfloat *param = state.param;   //�ⲿ����
	unsigned short dim = state.dim; //����ά��
	Derivefunc func = state.func;   //���ֺ���
	uint32_t index = 0;
	int s = 0;

	/* k1 */
	s = func(t, x0, k, param, _index);
	/*rk1*/
	//if (s != 0)
	//	return s;
	//for (index = 0; index < dim; index++)
	//{
	//	//x[index] += h / 6.0 * k[index];
	//	x0[index] += h*k[index];
	//}
	//return s;
	/*rk1*/
	if (s != 0)
		return s;
	for (index = 0; index < dim; index++)
	{
		x[index] = h / 6.0 * k[index];
		xtmp[index] = x0[index] + 0.5 * h * k[index];
	}

	/* k2 */
	s = func(t + 0.5 * h, xtmp, k, param, _index);
	if (s != 0)
		return s;
	for (index = 0; index < dim; index++)
	{
		x[index] += h / 3.0 * k[index];
		xtmp[index] = x0[index] + 0.5 * h * k[index];
	}

	/* k3 */
	s = func(t + 0.5 * h, xtmp, k, param, _index);
	if (s != 0)
		return s;
	for (index = 0; index < dim; index++)
	{
		x[index] += h / 3.0 * k[index];
		xtmp[index] = x0[index] + h * k[index];
	}

	/* k4*/
	s = func(t + h, xtmp, k, param, _index);
	if (s != 0)
		return s;
	for (index = 0; index < dim; index++)
	{
		x[index] += h / 6.0 * k[index];
		x0[index] += x[index];
	}
	return 0;
}

__device__ __host__ double getE(double _V, double _H)
{
	return 0.5 * _V * _V + g_0 * _H;
}

__device__ __host__ double getAlpha(double _speed, double _rangeAngle)
{
	return 5 + (_speed - SPEEDMIN) / (SPEEDMAX - SPEEDMIN) * _rangeAngle;
}

__device__ __host__ double getSigmaWithE(double _E, double _max, double _Edmax)
{
	if (fabs(_E) > ENERGINMAX)
		return _max;
	if (fabs(_E) < ENERGINMIN)
		return 0;
	if (_max > 0)
	{
		return (_E - SPEEDMIN) / (_Edmax - SPEEDMIN) * (_max);
	}
	else
	{
		return -((_E - SPEEDMIN) / (_Edmax - SPEEDMIN) * (-_max));
	}
}
__device__ __host__ double getSigma(double _speed, double _max, double _speedmax)
{
	if (fabs(_speed) > SPEEDMAX)
		return _max;
	if (fabs(_speed) < SPEEDMIN)
		return 0;
	if (_max > 0)
	{
		return (_speed - SPEEDMIN) / (_speedmax - SPEEDMIN) * (_max);
	}
	else
	{
		return -((_speed - SPEEDMIN) / (_speedmax - SPEEDMIN) * (-_max));
	}
}

//��ȡ�������
//Ŀ�꾭γ�� ��ǰ��γ�� Ԥ�⾭γ��
__device__ __host__ double getCrose(double _deslongt, double _deslat, double _srclongt, double _srclat, double _prelongt, double _prelat)
{
	spfloat tmp1 = lanuchAngle(_prelongt, _prelat, _srclongt, _srclat); //Ԥ��㺽���
	spfloat tmp2 = lanuchAngle(_deslongt, _deslat, _srclongt, _srclat); //Ŀ��㺽���
	spfloat dltapsi = tmp2 - tmp1;                                      //Ŀ�����Ԥ�����Ϊ��
	spfloat lengthc = length(_deslongt, _deslat, _srclongt, _srclat);
	return asin(sin(lengthc) * sin(dltapsi)); //��¼�������
}
